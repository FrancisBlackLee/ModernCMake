#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_lib.h"
#include <iostream>
using namespace std;

void GetCudaDeviceCount( int * num_gpus )
{
    hipGetDeviceCount( num_gpus );

    for ( int i = 0; i < *num_gpus; ++ i )
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("%d: %s\n", i, dprop.name);
    }
	cout << "Hello from cuda_lib.cu using std::cout\n";
}

